#include <stdio.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include "matmul.h"
using namespace std;

void allocateDeviceMemory(void** M, int size)
{
  hipError_t err = hipMalloc(M, size);
  assert(err==hipSuccess);
}

void deallocateDeviceMemory(void* M)
{
  hipError_t err = hipFree(M);
  assert(err==hipSuccess);
}

void matmul_ref(const int* const matrixA, const int* const matrixB,
                int* const matrixC, const int n) {
  // You can assume matrixC is initialized with zero
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      for (int k = 0; k < n; k++)
        matrixC[i * n + j] += matrixA[i * n + k] * matrixB[k * n + j];
}

void matmul_optimized(const int* const matrixA, const int* const matrixB,
                      int* const matrixC, const int* const d_A, const int* const d_B,  int* const d_C, const int n) {

  // TODO: Implement your CUDA code
}

